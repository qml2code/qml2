
#include <hip/hip_runtime.h>
double square(const double val) { return val * val; }

void fetch_square_distance_move_pointers(double *output, const double **q1,
                                         const double **q2,
                                         const long long nfeatures) {
  *output = 0.0;
  for (long long feature_id = 0; feature_id != nfeatures; ++feature_id) {
    *output += square(**q1 - **q2);
    (*q1)++;
    (*q2)++;
  }
}

void fetch_distance_move_pointers(double *output, const double **q1,
                                  const double **q2,
                                  const long long nfeatures) {
  fetch_square_distance_move_pointers(output, q1, q2, nfeatures);
  *output = sqrt(*output);
}
