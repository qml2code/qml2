#include "hip/hip_runtime.h"
void fetch_kernel_move_pointers(double *kernel_element, const double **q1,
                                const double **q2, double inv_sigma,
                                const long long nfeatures) {
  fetch_distance_move_pointers(kernel_element, q1, q2, nfeatures);
  *kernel_element = exp(-inv_sigma * *kernel_element);
}
